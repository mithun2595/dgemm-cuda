#include "hip/hip_runtime.h"
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "utils.h"
#include "types.h"
using namespace std;

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B) {

      const unsigned int TW = 32;
      const unsigned int TW1 = TW/4, TW2 = TW/2, TW3 = 3*TW/4;
      const unsigned int edge_limit = (int) ceilf((float)N/TW);
      const unsigned int ty = threadIdx.y, tx = threadIdx.x;
      const unsigned int by = blockIdx.y, bx = blockIdx.x;
      const unsigned int I = by*TW + ty, J = bx*TW + tx;

     __shared__ _DOUBLE_ As[TW][TW], Bs[TW][TW];

     _DOUBLE_ Cs[4] = {0};

    for(int kk = 0; kk < edge_limit; kk++)
    {

      if((kk*TW+tx)<N)
      {
        if(I+TW3 < N) {
          As[ty][tx] = A[I*N+(kk*TW+tx)];
          As[ty+TW1][tx] = A[(I+TW1)*N+(kk*TW+tx)];
          As[ty+TW2][tx] = A[(I+TW2)*N+(kk*TW+tx)];
          As[ty+TW3][tx] = A[(I+TW3)*N+(kk*TW+tx)];
        } else if(I+TW2 < N){
          As[ty][tx] = A[I*N+(kk*TW+tx)];
          As[ty+TW1][tx] = A[(I+TW1)*N+(kk*TW+tx)];
          As[ty+TW2][tx] = A[(I+TW2)*N+(kk*TW+tx)];
          As[ty+TW3][TW + tx] = 0;
        } else if(I+TW1 < N) {
          As[ty][tx] = A[I*N+(kk*TW+tx)];
          As[ty+TW1][tx] = A[(I+TW1)*N+(kk*TW+tx)];
          As[ty+TW2][tx] = 0;
          As[ty+TW3][TW + tx] = 0;
        } else if(I < N) {
          As[ty][tx] = A[I*N+(kk*TW+tx)];
          As[ty+TW1][tx] = 0;
          As[ty+TW2][tx] = 0;
          As[ty+TW3][tx] = 0;
        } else {
          As[ty][tx] = 0;
          As[ty+TW1][tx] = 0;
          As[ty+TW2][tx] = 0;
          As[ty+TW3][tx] = 0;
        }

      } else {
        As[ty][tx] = 0;
        As[ty+TW1][tx] = 0;
        As[ty+TW2][tx] = 0;
        As[ty+TW3][tx] = 0;
      }

      if(J<N)
      {
        if(kk*TW+ty+TW3 < N) {
          Bs[ty][tx] = B[(kk*TW+ty)*N+J];
          Bs[ty+TW1][tx] = B[(kk*TW+ty+TW1)*N+J];
          Bs[ty+TW2][tx] = B[(kk*TW+ty+TW2)*N+J];
          Bs[ty+TW3][tx] = B[(kk*TW+ty+TW3)*N+J];
        } else if(kk*TW+ty+TW2 < N){
          Bs[ty][tx] = B[(kk*TW+ty)*N+J];
          Bs[ty+TW1][tx] = B[(kk*TW+ty+TW1)*N+J];
          Bs[ty+TW2][tx] = B[(kk*TW+ty+TW2)*N+J];
          Bs[ty+TW3][tx] = 0;
        } else if(kk*TW+ty+TW1 < N) {
          Bs[ty][tx] = B[(kk*TW+ty)*N+J];
          Bs[ty+TW1][tx] = B[(kk*TW+ty+TW1)*N+J];
          Bs[ty+TW2][tx] = 0;
          Bs[ty+TW3][tx] = 0;
        } else if(kk*TW+ty < N) {
          Bs[ty][tx] = B[(kk*TW+ty)*N+J];
          Bs[ty+TW1][tx] = 0;
          Bs[ty+TW2][tx] = 0;
          Bs[ty+TW3][tx] = 0;
        } else {
          Bs[ty][tx] = 0;
          Bs[ty+TW1][tx] = 0;
          Bs[ty+TW2][tx] = 0;
          Bs[ty+TW3][tx] = 0;
        }
      } else {
        Bs[ty][tx] = 0;
        Bs[ty+TW1][tx] = 0;
        Bs[ty+TW2][tx] = 0;
        Bs[ty+TW3][tx] = 0;
      }
      __syncthreads();

      #pragma unroll
      for(int k = 0; k < TW; k++)
      {
        Cs[0] += As[ty][k] * Bs[k][tx];
        Cs[1] += As[ty+TW1][k] * Bs[k][tx];
        Cs[2] += As[ty+TW2][k] * Bs[k][tx];
        Cs[3] += As[ty+TW3][k] * Bs[k][tx];
      }

      __syncthreads();
    }
    if((J<N))
    {
      if(I+TW3 < N) {
        C[I*N+J] = Cs[0];
        C[(I+TW1)*N+J] = Cs[1];
        C[(I+TW2)*N+J] = Cs[2];
        C[(I+TW3)*N+J] = Cs[3];
      } else if(I+TW2 < N){
        C[I*N+J] = Cs[0];
        C[(I+TW1)*N+J] = Cs[1];
        C[(I+TW2)*N+J] = Cs[2];
      } else if(I+TW1 < N) {
        C[I*N+J] = Cs[0];
        C[(I+TW1)*N+J] = Cs[1];
      } else if(I < N) {
        C[I*N+J] = Cs[0];
      }
    }
}
